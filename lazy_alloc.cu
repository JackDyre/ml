#ifndef LAZY_ALLOC_CU
#define LAZY_ALLOC_CU

#include <cassert>
#include <cstdlib>
#include <ctime>

typedef enum DeviceState {
  /*
   * The ptr is owned and the data is
   * on the current device
   */
  OWNED_VALID,
  /*
   * The ptr is borrowed and the data is
   * on the current device
   */
  BORROWED_VALID,
  /*
   * The ptr is owned and the data is
   * not on the current device
   */
  OWNED_INVALID,
  /*
   * The ptr is borrowed and the data is
   * not on the current device
   */
  BORROWED_INVALID,
  /*
   * We do not have any allocated memory
   * on the current device
   */
  NO_ALLOC,
} DeviceState;

/*
 * Helper class that lazily manages memory on the
 * host and device and provides a unified interface
 * for using the same data in both locations
 */
class LazyDeviceAllocator {
private:
  size_t alloc_size;

  void *host_ptr = NULL;
  DeviceState host_state = NO_ALLOC;

  void *dev_ptr = NULL;
  DeviceState dev_state = NO_ALLOC;

  LazyDeviceAllocator() {}

public:
  void set_host_valid_unchecked() {
    if (host_state == OWNED_INVALID) {
      host_state = OWNED_VALID;
    } else if (host_state == BORROWED_INVALID) {
      host_state = BORROWED_VALID;
    }
  }

  void set_host_invalid_unchecked() {
    if (host_state == OWNED_VALID) {
      host_state = OWNED_INVALID;
    } else if (host_state == BORROWED_VALID) {
      host_state = BORROWED_INVALID;
    }
  }

  void set_dev_valid_unchecked() {
    if (dev_state == OWNED_INVALID) {
      dev_state = OWNED_VALID;
    } else if (dev_state == BORROWED_INVALID) {
      dev_state = BORROWED_VALID;
    }
  }

  void set_dev_invalid_unchecked() {
    if (dev_state == OWNED_VALID) {
      dev_state = OWNED_INVALID;
    } else if (dev_state == BORROWED_VALID) {
      dev_state = BORROWED_INVALID;
    }
  }

  void set_host_state_unchecked(DeviceState state) { host_state = state; }
  void set_dev_state_unchecked(DeviceState state) { dev_state = state; }

  LazyDeviceAllocator &operator=(const LazyDeviceAllocator &) = delete;

  ~LazyDeviceAllocator() {
    free_host();
    free_dev();
  }

  static LazyDeviceAllocator new_no_alloc(size_t alloc_size) {
    LazyDeviceAllocator d;
    d.set_alloc_size_unchecked(alloc_size);
    return d;
  }

  static LazyDeviceAllocator new_owned_host(void *host_ptr, size_t size) {
    auto d = LazyDeviceAllocator::new_no_alloc(size);
    assert(host_ptr != NULL);
    d.set_host_ptr_unchecked(host_ptr);
    d.host_state = OWNED_VALID;
    return d;
  }

  static LazyDeviceAllocator new_borrowed_host(void *host_ptr, size_t size) {
    auto d = LazyDeviceAllocator::new_no_alloc(size);
    assert(host_ptr != NULL);
    d.set_host_ptr_unchecked(host_ptr);
    d.host_state = BORROWED_VALID;
    return d;
  }

  static LazyDeviceAllocator new_owned_dev(void *dev_ptr, size_t size) {
    auto d = LazyDeviceAllocator::new_no_alloc(size);
    assert(dev_ptr != NULL);
    d.set_dev_ptr_unchecked(dev_ptr);
    d.dev_state = OWNED_VALID;
    return d;
  }

  static LazyDeviceAllocator new_borrowed_dev(void *dev_ptr, size_t size) {
    auto d = LazyDeviceAllocator::new_no_alloc(size);
    assert(dev_ptr != NULL);
    d.set_dev_ptr_unchecked(dev_ptr);
    d.dev_state = BORROWED_VALID;
    return d;
  }

  void ensure_on_host() {
    ensure_host_alloced();

    if (host_is_valid()) {
      // Host is already valid, just return
      return;

    } else if (dev_is_valid()) {
      // We have a valid device ptr to copy
      // from, and an alloced but invalid
      // host ptr to copy to.
      copy_to_host_unchecked();
      set_host_valid_unchecked();
      set_dev_invalid_unchecked();
      return;

    } else {
      // We don't have a valid device ptr to
      // copy from, so we just label the host
      // as valid and return
      set_host_valid_unchecked();
      return;
    }
  }

  void ensure_on_dev() {
    ensure_dev_alloced();

    if (dev_is_valid()) {
      // Device is already valid, just return
      return;

    } else if (host_is_valid()) {
      // We have a valid host ptr to copy
      // from, and an alloced but invalid
      // device ptr to copy to.
      copy_to_dev_unchecked();
      set_dev_valid_unchecked();
      set_host_invalid_unchecked();
      return;

    } else {
      // We don't have a valid device ptr to
      // copy from, so we just label the host
      // as valid and return
      set_dev_valid_unchecked();
      return;
    }
  }

  /*
   * NOTE: The caller is responseible for ensuring
   *       that the host data is not overwritten,
   *       and that both ptrs are allocated. This
   *       method does not set the DeviceState
   *       flags
   */
  void copy_to_host_unchecked() {
    assert(hipSuccess ==
           hipMemcpy(host_ptr, dev_ptr, alloc_size, hipMemcpyDeviceToHost));
  }

  /*
   * NOTE: The caller is responseible for ensuring
   *       that the device data is not overwritten,
   *       and that both ptrs are allocated. This
   *       method does not set the DeviceState
   *       flags
   */
  void copy_to_dev_unchecked() {
    assert(hipSuccess ==
           hipMemcpy(dev_ptr, host_ptr, alloc_size, hipMemcpyHostToDevice));
  }

  /*
   * NOTE: The caller is responsible for ensuring the
   *       overwritten ptr is not leaked. This method
   *       does not set DeviceState flags
   */
  void alloc_host_unchecked() {
    void *ptr = (float *)std::malloc(alloc_size);
    assert(ptr != NULL);
    set_host_ptr_unchecked(ptr);
  }

  void ensure_host_alloced() {
    if (host_state == NO_ALLOC) {
      alloc_host_unchecked();
      host_state = OWNED_INVALID;
    }
  }

  /*
   * NOTE: The caller is responsible for ensuring the
   *       overwritten ptr is not leaked. This method
   *       does not set DeviceState flags
   */
  void alloc_dev_unchecked() {
    void *ptr;
    auto err = hipMalloc(&ptr, alloc_size);
    assert(err == hipSuccess);
    set_dev_ptr_unchecked(ptr);
  }

  void ensure_dev_alloced() {
    if (dev_state == NO_ALLOC) {
      alloc_dev_unchecked();
      dev_state = OWNED_INVALID;
    }
  }

  void free_host() {
    if (host_state == OWNED_VALID || host_state == OWNED_INVALID) {
      free(host_ptr);
    }
    host_ptr = NULL;
    host_state = NO_ALLOC;
  }

  void free_dev() {
    if (dev_state == OWNED_VALID || dev_state == OWNED_INVALID) {
      assert(hipSuccess == hipFree(dev_ptr));
    }
    dev_ptr = NULL;
    dev_state = NO_ALLOC;
  }

  bool host_is_valid() {
    return host_state == OWNED_VALID || host_state == BORROWED_VALID;
  }

  bool dev_is_valid() {
    return dev_state == OWNED_VALID || dev_state == BORROWED_VALID;
  }

  /*
   * NOTE: The caller is responsible for ensuring the
   *       alloc size is not changed while any of the
   *       pointers are valid
   */
  void set_alloc_size_unchecked(size_t alloc_size) {
    this->alloc_size = alloc_size;
  }

  /*
   * NOTE: The caller is responsible for ensuring the
   *       ptr is the same size as the alloc size and
   *       that the overwritten ptr is not leaked. This
   *       method does not set DeviceState flags
   */
  void set_host_ptr_unchecked(void *host_ptr) { this->host_ptr = host_ptr; }

  /*
   * NOTE: The caller is responsible for ensuring the
   *       ptr is valid
   */
  void *get_host_ptr_unchecked() { return host_ptr; }

  void *get_host_ptr() {
    ensure_on_host();
    return host_ptr;
  }

  /*
   * NOTE: The caller is responsible for ensuring the
   *       ptr is the same size as the alloc size and
   *       that the overwritten ptr is not leaked. This
   *       method does not set DeviceState flags
   */
  void set_dev_ptr_unchecked(void *dev_ptr) { this->dev_ptr = dev_ptr; }

  /*
   * NOTE: The caller is responsible for ensuring the
   *       ptr is valid
   */
  void *get_dev_ptr_unchecked() { return dev_ptr; }

  void *get_dev_ptr() {
    ensure_on_dev();
    return dev_ptr;
  }

  size_t get_alloc_size() { return alloc_size; }
};

#endif // LAZY_ALLOC_CU

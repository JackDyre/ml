#include "hip/hip_runtime.h"
#include "kernels.h"
#include "util.h"
#include <cassert>
#include <cstddef>
#include <hiprand/hiprand_kernel.h>

#define thread_block_idx(dim) blockDim.dim *blockIdx.dim + threadIdx.dim
#define panic_on_cuda_error(err) assert(err == hipSuccess)

const auto THREADS_PER_BLOCK = dim3(16, 16, 16);

#define block_shape_dim(launch_shape, dim)                                     \
  (launch_shape.dim + THREADS_PER_BLOCK.dim - 1) / THREADS_PER_BLOCK.dim

#define kernel_config(x, y, z) THREADS_PER_BLOCK, blocks_per_grid(dim3(x, y, z))

dim3 blocks_per_grid(dim3 launch_shape) {
  return dim3(block_shape_dim(launch_shape, x),
              block_shape_dim(launch_shape, y),
              block_shape_dim(launch_shape, z));
}

__global__ void kernel_matrix_fill(MatrixFill args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.rows || c >= args.cols || z != 0) {
    return;
  }

  args.ptr[mat_idx(r, c, args.stride)] = args.val;
}

void device_matrix_fill(MatrixFill args) {
  kernel_matrix_fill<<<kernel_config(args.rows, args.cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

__global__ void kernel_matrix_rand(MatrixRand args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.rows || c >= args.cols || z != 0) {
    return;
  }

  hiprandState state;
  hiprand_init(args.seed, mat_idx(r, c, args.stride), 0, &state);

  args.ptr[mat_idx(r, c, args.stride)] =
      args.low + (args.high - args.low) * hiprand_uniform(&state);
}

void device_matrix_rand(MatrixRand args) {
  kernel_matrix_rand<<<kernel_config(args.rows, args.cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

__global__ void kernel_matrix_add(MatrixAdd args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.rows || c >= args.cols || z != 0) {
    return;
  }

  auto other_val = args.other_ptr[mat_idx(r, c, args.other_stride)];

  args.dst_ptr[mat_idx(r, c, args.dst_stride)] += other_val;
}

void device_matrix_add(MatrixAdd args) {
  kernel_matrix_add<<<kernel_config(args.rows, args.cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

__global__ void kernel_matrix_mul(MatrixMul args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.dst_rows || c >= args.dst_cols || z != 0) {
    return;
  }

  float val = 0.0f;
  for (std::size_t i = 0; i < args.inner_dim; i++) {
    float left_val = args.l_ptr[mat_idx(r, i, args.l_stride)];
    float right_val = args.r_ptr[mat_idx(i, c, args.r_stride)];
    val += left_val * right_val;
  }
  args.dst_ptr[mat_idx(r, c, args.dst_stride)] = val;
}

void device_matrix_mul(MatrixMul args) {
  kernel_matrix_mul<<<kernel_config(args.dst_rows, args.dst_cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

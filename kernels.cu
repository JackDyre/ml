#include "hip/hip_runtime.h"
#include "kernel_types.h"
#include "kernels.h"
#include "util.h"
#include <cassert>
#include <cstddef>
#include <hiprand/hiprand_kernel.h>

#define thread_block_idx(dim) blockDim.dim *blockIdx.dim + threadIdx.dim

const auto THREADS_PER_BLOCK = dim3(16, 16, 16);

#define block_shape_dim(launch_shape, dim)                                     \
  (launch_shape.dim + THREADS_PER_BLOCK.dim - 1) / THREADS_PER_BLOCK.dim

#define kernel_config(x, y, z) THREADS_PER_BLOCK, blocks_per_grid(dim3(x, y, z))

dim3 blocks_per_grid(dim3 launch_shape) {
  return dim3(block_shape_dim(launch_shape, x),
              block_shape_dim(launch_shape, y),
              block_shape_dim(launch_shape, z));
}

__global__ void kernel_matrix_fill(MatrixFill args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.shape.rows || c >= args.shape.cols || z != 0) {
    return;
  }

  args.ptr[mat_idx_spec(r, c, args.idx_spec)] = args.val;
}

void device_matrix_fill(MatrixFill args) {
  auto l_rows = args.shape.rows;
  auto l_cols = args.shape.cols;
  kernel_matrix_fill<<<kernel_config(l_rows, l_cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

__global__ void kernel_matrix_rand(MatrixRand args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.shape.rows || c >= args.shape.cols || z != 0) {
    return;
  }

  auto idx = mat_idx_spec(r, c, args.idx_spec);

  hiprandState state;
  hiprand_init(args.seed, idx, 0, &state);

  args.ptr[idx] = args.low + (args.high - args.low) * hiprand_uniform(&state);
}

void device_matrix_rand(MatrixRand args) {
  auto l_rows = args.shape.rows;
  auto l_cols = args.shape.cols;
  kernel_matrix_rand<<<kernel_config(l_rows, l_cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

__global__ void kernel_matrix_add(MatrixAdd args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.shape.rows || c >= args.shape.cols || z != 0) {
    return;
  }

  auto l_val = args.l_ptr[mat_idx_spec(r, c, args.l_idx_spec)];
  auto r_val = args.r_ptr[mat_idx_spec(r, c, args.r_idx_spec)];

  args.dst_ptr[mat_idx_spec(r, c, args.dst_idx_spec)] = l_val + r_val;
}

void device_matrix_add(MatrixAdd args) {
  auto l_rows = args.shape.rows;
  auto l_cols = args.shape.cols;
  kernel_matrix_add<<<kernel_config(l_rows, l_cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

__global__ void kernel_matrix_mul(MatrixMul args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.shape.rows || c >= args.shape.cols || z != 0) {
    return;
  }

  float val = 0.0f;
  for (std::size_t i = 0; i < args.inner_dim; i++) {
    float left_val = args.l_ptr[mat_idx_spec(r, i, args.l_idx_spec)];
    float right_val = args.r_ptr[mat_idx_spec(i, c, args.r_idx_spec)];
    val += left_val * right_val;
  }
  args.dst_ptr[mat_idx_spec(r, c, args.dst_idx_spec)] = val;
}

void device_matrix_mul(MatrixMul args) {
  auto l_rows = args.shape.rows;
  auto l_cols = args.shape.cols;
  kernel_matrix_mul<<<kernel_config(l_rows, l_cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

__global__ void kernel_matrix_relu(MatrixRelu args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.shape.rows || c >= args.shape.cols || z != 0) {
    return;
  }

  args.dst_ptr[mat_idx_spec(r, c, args.dst_idx_spec)] =
      relu(args.src_ptr[mat_idx_spec(r, c, args.src_idx_spec)]);
}

void device_matrix_relu(MatrixRelu args) {
  auto l_rows = args.shape.rows;
  auto l_cols = args.shape.cols;
  kernel_matrix_relu<<<kernel_config(l_rows, l_cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

__global__ void kernel_matrix_se(MatrixSE args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.shape.rows || c >= args.shape.cols || z != 0) {
    return;
  }

  auto diff = args.a_ptr[mat_idx_spec(r, c, args.a_idx_spec)] -
              args.b_ptr[mat_idx_spec(r, c, args.b_idx_spec)];

  args.dst_ptr[mat_idx_spec(r, c, args.dst_idx_spec)] = diff * diff;
}

void device_matrix_se(MatrixSE args) {
  auto l_rows = args.shape.rows;
  auto l_cols = args.shape.cols;
  kernel_matrix_se<<<kernel_config(l_rows, l_cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}

__global__ void kernel_matrix_relu_deriv(MatrixReluDeriv args) {
  std::size_t r = thread_block_idx(x);
  std::size_t c = thread_block_idx(y);
  std::size_t z = thread_block_idx(z);

  if (r >= args.shape.rows || c >= args.shape.cols || z != 0) {
    return;
  }

  args.dst_ptr[mat_idx_spec(r, c, args.dst_idx_spec)] =
      relu_deriv(args.src_ptr[mat_idx_spec(r, c, args.src_idx_spec)]);
}

void device_matrix_relu_deriv(MatrixReluDeriv args) {
  auto l_rows = args.shape.rows;
  auto l_cols = args.shape.cols;
  kernel_matrix_relu_deriv<<<kernel_config(l_rows, l_cols, 1)>>>(args);
  auto err = hipDeviceSynchronize();
  panic_on_cuda_error(err);
}


#include <hip/hip_runtime.h>
#include <iostream>
int main(void) {
  hipDeviceProp_t prop;
  auto err = hipGetDeviceProperties(&prop, 0);
  if (err != hipSuccess) {
    return 1;
  }
  std::cout << "sm_" << prop.major << prop.minor;
}
